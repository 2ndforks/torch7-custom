#include <hipblas.h>
//#include "hipsparse.h"

static int cunxn_testSgemm_run(lua_State *L)
{
  //THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *A = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *B = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *C = (THCudaTensor *)luaT_checkudata(L, 4, "torch.CudaTensor");

  long m = luaT_getfieldcheckint(L, 1, "m");
  long n = luaT_getfieldcheckint(L, 1, "n");
  long k = luaT_getfieldcheckint(L, 1, "k");

  long lda = luaT_getfieldcheckint(L, 1, "lda");
  long ldb = luaT_getfieldcheckint(L, 1, "ldb");
  long ldc = luaT_getfieldcheckint(L, 1, "ldc");

  long tA = luaT_getfieldcheckint(L, 1, "tA");
  long tB = luaT_getfieldcheckint(L, 1, "tB");

  hipblasOperation_t transa=HIPBLAS_OP_N;
  hipblasOperation_t transb=HIPBLAS_OP_N;


	if(tA==1) { transa=HIPBLAS_OP_T; }
	if(tB==1) { transb=HIPBLAS_OP_T; }

  float alpha = 1.0f;
  float beta = 1.0f;


  float* ptrA  = THCudaTensor_data(A);
  float* ptrB   = THCudaTensor_data(B);
  float* ptrC    = THCudaTensor_data(C);

	hipblasHandle_t handle;


  hipblasStatus_t err = hipblasCreate(&handle);
  if (err != HIPBLAS_STATUS_SUCCESS) {
    printf("error in creating handle");
  }

printf("calling sgemm...");
  err = hipblasSgemm(handle,
                           transa, transb,
                           m, n, k,
                           &alpha,
                           ptrA, lda,
                           ptrB, ldb,
                           &beta,
                           ptrC, ldc);
  if (err != HIPBLAS_STATUS_SUCCESS) {
    printf("error in sgemm");
  }
printf("called sgemm...");

err = hipblasDestroy(handle);
  if (err != HIPBLAS_STATUS_SUCCESS) {
    printf("error in destroying handle");
  }



  return 1;
}

#if 0
static int cunxn_testSgemm_cusparserun(lua_State *L)
{

  THCudaTensor *output = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THIntTensor *coorow = (THIntTensor *)luaT_checkudata(L, 3, "torch.IntTensor");
  THIntTensor *coocol = (THIntTensor *)luaT_checkudata(L, 4, "torch.IntTensor");
  THCudaTensor *cooval = (THCudaTensor *)luaT_checkudata(L, 5, "torch.CudaTensor");
  THCudaTensor *w      = (THCudaTensor *)luaT_checkudata(L, 6, "torch.CudaTensor");

  hipsparseHandle_t handle =0;

  hipsparseStatus_t err = hipsparseCreate(&handle);
  if (err != HIPSPARSE_STATUS_SUCCESS) {
    printf("error in creating handle");
  }

   
   // convert coorow to ints :
   
   float* csrValA   = THCudaTensor_data(cooval);

   float* wptr      = THCudaTensor_data(w);
   int m =  100000;
   int n = 2075;
   int k = 132809;
   
   int ldc = k;
   int ldb = 100000;

   float alpha=1;
   float beta=0;
   
   int sz=coorow->size[1];
   int nnz=sz;
   
   printf("nnz : %d\n", nnz);
   printf("ldb : %d\n", ldb);
   printf("step 1");
     
   int* coorowHptr  = THIntTensor_data(coorow);
   int* coocolHptr  = THIntTensor_data(coocol);
   int* csrColInd =0;
   hipMalloc((void**)&csrColInd, sizeof(int) * sz);
   hipMemcpy(csrColInd, coocolHptr, sizeof(int) * sz, hipMemcpyHostToDevice);

   printf("step 2");
   
   int* cooRowInd =0;
   int* csrRowPtr =0;
   hipMalloc((void**)&cooRowInd, sizeof(int) * sz);
   hipMemcpy(cooRowInd, coorowHptr, sizeof(int) * sz, hipMemcpyHostToDevice);
   err = hipsparseXcoo2csr(handle, cooRowInd, sz, m, csrRowPtr, HIPSPARSE_INDEX_BASE_ONE);
   hipFree(cooRowInd);

   printf("step 3");

   hipsparseMatDescr_t descrA;

   err = hipsparseCreateMatDescr(&descrA);
   err = hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);
   if (err != HIPSPARSE_STATUS_SUCCESS) {
    printf("error in descriptor");
   }
  
  hipsparseOperation_t trans = HIPSPARSE_OPERATION_TRANSPOSE;

   printf("step 4");
  
  //THCudaTensor_resize2d(output, ldb, m);
  //THCudaTensor_fill(output,0);
  float* optr = THCudaTensor_data(output);
  //int ldc=ldb;

   printf("step 5");
  
   err = hipsparseScsrmm( handle, trans, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtr, csrColInd, wptr, ldb, &beta, optr, ldc);

   printf("step 6");


   if (err != HIPSPARSE_STATUS_SUCCESS) {
    printf("error in csrmm");
   }

   if (err == HIPSPARSE_STATUS_NOT_INITIALIZED) {
    printf("HIPSPARSE_STATUS_NOT_INITIALIZED");
   }
               
   if (err == HIPSPARSE_STATUS_ALLOC_FAILED) {
    printf("HIPSPARSE_STATUS_ALLOC_FAILED");
   }
               
   if (err == HIPSPARSE_STATUS_INVALID_VALUE) {
    printf("HIPSPARSE_STATUS_INVALID_VALUE");
   }
               
   if (err == HIPSPARSE_STATUS_ARCH_MISMATCH) {
    printf("HIPSPARSE_STATUS_ARCH_MISMATCH");
   }
               
   if (err == HIPSPARSE_STATUS_EXECUTION_FAILED) {
    printf("HIPSPARSE_STATUS_EXECUTION_FAILED");
   }
               
   if (err == HIPSPARSE_STATUS_INTERNAL_ERROR) {
    printf("HIPSPARSE_STATUS_INTERNAL_ERROR");
   }
               
   if (err == HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED) {
    printf("HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED");
   }
               
  err = hipsparseDestroy(handle);
  if (err != HIPSPARSE_STATUS_SUCCESS) {
    printf("error in destroying handle");
  }
return 0;
}
#endif

static const struct luaL_Reg cunxn_testSgemm__ [] = {
  {"testSgemm_run", cunxn_testSgemm_run},
//  {"testSgemm_cusparserun", cunxn_testSgemm_cusparserun},
  {NULL, NULL}
};

static void cunxn_testSgemm_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunxn_testSgemm__, "nxn");
  lua_pop(L,1);
}
