#include "hip/hip_runtime.h"
#ifndef assert
#define assert(e)  \
    if (!(e)) { \
        printf("failed assertion `%s'\n", #e); \
        THError("aborting..."); \
    };
#endif


__global__ void copyPixelsInSlices(float *ptrinput, float *ptrkslices,
	int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int valuesperthread)
{
	const int pixi=blockIdx.x;
	const int pixj=blockIdx.y;
	const int blk =blockDim.x;
	const int tidx=threadIdx.x;

        int imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
        int jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
        int imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
        int jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;

	int i;
	int j;
	int k;

	ptrinput   += (pixi * isize2 + pixj) * nInputPlane ;
	ptrkslices += ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane;

	int stridej = (kH*kW - dW) * nInputPlane;
	int stridei = (((size2-jmax+jmin-1)*kH -dH)*kW  + (jmax-jmin+1)*dW)*nInputPlane;
	
	for(i=imin; i<imax+1; i++) {
		for(j=jmin; j<jmax+1; j++) {
			for(k=0; k<valuesperthread; k++) {
				ptrkslices[k*blk+tidx]=ptrinput[k*blk+tidx];
			}
			ptrkslices += stridej;
		}
		ptrkslices += stridei;
	}	
}


template <int maxnumplanes> __global__ void copyPixelsInSlicesSharedMem(float *ptrinput, float *ptrkslices,
	int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int valuesperthread)
{
	const int pixi=blockIdx.x;
	const int pixj=blockIdx.y;
	const int blk =blockDim.x;
	const int tidx=threadIdx.x;

	__shared__ float pixvalues[maxnumplanes];

        int imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
        int jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
        int imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
        int jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;

	int i;
	int j;
	int k;

	ptrinput   += (pixi * isize2 + pixj) * nInputPlane ;
	ptrkslices += ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane;

	int stridej = (kH*kW - dW) * nInputPlane;
	int stridei = (((size2-jmax+jmin-1)*kH -dH)*kW  + (jmax-jmin+1)*dW)*nInputPlane;


	// load the stuff in shared memory first...
	for(k=0; k<valuesperthread; k++) {
		pixvalues[k*blk+tidx]=ptrinput[k*blk+tidx];
	}
	

	
	for(i=imin; i<imax+1; i++) {
		for(j=jmin; j<jmax+1; j++) {
			for(k=0; k<valuesperthread; k++) {
				ptrkslices[k*blk+tidx]=pixvalues[k*blk+tidx];
			}
			ptrkslices += stridej;
		}
		ptrkslices += stridei;
	}	
}


template <int maxnumplanes> __global__ void copyBiasToOutputsSharedMem(float *ptrbias, float *ptroutput, int size1, int size2, int nOutputPlane, int valuesperthread)
{
	// each thread has a value to manage...
	const int blk =blockDim.x;
	const int tidx=threadIdx.x;

	__shared__ float biasvalues[maxnumplanes];
 
	int i;
	int j;
	int k;

	// load all in shared mem
	for(k=0; k<valuesperthread;k++) {
		biasvalues[k*blk+tidx]=ptrbias[k*blk+tidx];
	}

	// copy to output
	for(i=0; i<size1; i++) {
		for(j=0; j<size2; j++) {
			for(k=0; k<valuesperthread;k++) {
				ptroutput[k*blk+tidx]=biasvalues[k*blk+tidx];
			}
			ptroutput+=nOutputPlane;
		}
	}
}


__global__ void copyBiasToOutputs(float *ptrbias, float *ptroutput, int size1, int size2, int nOutputPlane, int valuesperthread)
{
	// each thread has a value to manage...
	const int blk =blockDim.x;
	const int tidx=threadIdx.x;

	int i;
	int j;
	int k;

	for(i=0; i<size1; i++) {
		for(j=0; j<size2; j++) {
			for(k=0; k<valuesperthread; k++) {
				ptroutput[k*blk+tidx]=ptrbias[k*blk+tidx];
			}
			ptroutput+=nOutputPlane;
		}
	}
}




static int cunn_SpatialConvolutionNew_updateOutput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *kernels = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *kslicestest = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "kslicestest", "torch.CudaTensor");
  long kW = luaT_getfieldcheckint(L, 1, "kW");
  long kH = luaT_getfieldcheckint(L, 1, "kH");
  long dW = luaT_getfieldcheckint(L, 1, "dW");
  long dH = luaT_getfieldcheckint(L, 1, "dH");
  long shdmem = luaT_getfieldcheckint(L, 1, "shdmem");
  long nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  long nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");

  //luaL_argcheck(L, dimension >= 0 && dimension < input->nDimension, 2, "dimension out of range");

  assert(nInputPlane%32 == 0);
  assert(nOutputPlane%32 == 0);

  // unfold the input tensor 
  // input should be contiguous already but... well.
  input = THCudaTensor_newContiguous(input);

  // find the size of kernelslices
  long isize1 = input->size[0];
  long isize2 = input->size[1];
  long size1 = (isize1 - kH) / dH + 1;
  long size2 = (isize2 - kW) / dW + 1;

  THCudaTensor* kernelSlices = THCudaTensor_newWithSize1d(size1*size2*kW*kH*nInputPlane);
  THCudaTensor_resize2d(output, size1* size2, nOutputPlane);

  float* ptrkslices = THCudaTensor_data(kernelSlices);
  float* ptroutput  = THCudaTensor_data(output);
  float* ptrinput   = THCudaTensor_data(input);
  float* ptrbias    = THCudaTensor_data(bias);

  // cuda blocks & threads:
  dim3 blocks (isize1, isize2);
  dim3 threads (32);
  long valuesperthread=nInputPlane/32;

  
  //kernel unfold inputs
  if (nInputPlane >1024 || shdmem==0) {
  copyPixelsInSlices<<<blocks, threads>>>(ptrinput, ptrkslices,
	dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread);
  }
  else if (nInputPlane >512) {
        printf("using shared memory 1024 floats\n");
        copyPixelsInSlicesSharedMem <1024> <<<blocks, threads>>>(ptrinput, ptrkslices,
	dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread);
  }
  else if (nInputPlane >384) {
        printf("using shared memory 512 floats\n");
        copyPixelsInSlicesSharedMem <512> <<<blocks, threads>>>(ptrinput, ptrkslices,
	dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread);
  }
  else if (nInputPlane >256) {
        printf("using shared memory 384 floats\n");
        copyPixelsInSlicesSharedMem <384> <<<blocks, threads>>>(ptrinput, ptrkslices,
	dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread);
  }
  else if (nInputPlane >128) {
        printf("using shared memory 256 floats\n");
        copyPixelsInSlicesSharedMem <256> <<<blocks, threads>>>(ptrinput, ptrkslices,
	dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread);
  }
  else {
        printf("using shared memory 128 floats\n");
        copyPixelsInSlicesSharedMem <128> <<<blocks, threads>>>(ptrinput, ptrkslices,
	dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread);
  }
  


  // fill output with biases : use shared memory 48K if possible, there is only one block anyway
  dim3 blocksbias (1);
  dim3 threadsbias (32);

  if (nOutputPlane >12256 || shdmem==0) {
  	copyBiasToOutputs<<<blocksbias, threadsbias>>>(ptrbias, ptroutput, size1, size2, nOutputPlane, nOutputPlane/32);  }
  else {
        printf("using shared memory 12k floats\n");
  	copyBiasToOutputsSharedMem<12256><<<blocksbias, threadsbias>>>(ptrbias, ptroutput, size1, size2, nOutputPlane, nOutputPlane/32);  }


  // unfold conv kernels by resizing
  THCudaTensor_resize2d(kernels, nOutputPlane, kW*kH*nInputPlane);
  THCudaTensor_transpose(kernels, NULL, 0, 1);
  // put kernelslices in matrix mode
  THCudaTensor_resize2d(kernelSlices, size1*size2,kW*kH*nInputPlane);


  // do addmm on output
  
  THCudaTensor_addmm(output, 1,1, kernelSlices, kernels);


  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in copyPixelsInSlices: %s\n", hipGetErrorString(err));
    THError("aborting");
  }

  THCudaTensor_resize3d(output, size1, size2, nOutputPlane);
 
  THCudaTensor_resizeAs(kslicestest, kernelSlices);
  THCudaTensor_copy(kslicestest, kernelSlices);

  // final cut:
  THCudaTensor_free(input); 
  THCudaTensor_free(kernelSlices); 
  //THCudaTensor_select(output, NULL, dimension, 0);

  return 1;
}





static int cunn_SpatialConvolutionNew_updateGradInput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");

  luaL_argcheck(L, dW == 1, 1, "dW must be 1 (this is only a limit for CudaTensors)");
  luaL_argcheck(L, dH == 1, 1, "dH must be 1 (this is only a limit for CudaTensors)");

  THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  if (input->nDimension == 3)
  {
    /* check dims */
    THArgCheck(nOutputPlane == gradOutput->size[0], 1, "Number of output features is not equal to nOutputPlane");

    /* gradient to input */
    THCudaTensor *tweight = THCudaTensor_newTranspose(weight,0,1);
    THCudaTensor_conv2Dmv(gradInput, 0.0, gradOutput, tweight, dH, dW, "fc");
    THCudaTensor_free(tweight);
  }
  else 
  {
    /* check dims */
    THArgCheck(nOutputPlane == gradOutput->size[1], 1, "Number of output features is not equal to nOutputPlane");

    /* gradient to input */
    THCudaTensor *tweight = THCudaTensor_newTranspose(weight,0,1);
    THCudaTensor_conv2Dmm(gradInput, 0.0, gradOutput, tweight, dH, dW, "fc");
    THCudaTensor_free(tweight);    
  }

  return 1;
}

__global__ void compute_gradBias1(float *gradBias, float *gradOutput, float scale,
                                 int output_n, int output_h, int output_w)
{
  // each block does a plane
  int k = blockIdx.x;
  float *gradOutput_k = gradOutput + (k + threadIdx.y*output_n)*output_h*output_w;

  // offsets
  int i_start = threadIdx.x;
  int i_end = output_w*output_h;
  int i_step = blockDim.x;

  int tid = threadIdx.x + threadIdx.y * blockDim.x;
  int nthreads = blockDim.x * blockDim.y;

  // sum output plane k into partial sum array
  __shared__ float sums[512];
  sums[tid] = 0;
  for (int i=i_start; i<i_end; i+=i_step) {
    sums[tid] += gradOutput_k[i];
  }
  __syncthreads();

  // reduce
  if (tid == 0) {
    for (int i=0; i<nthreads; i++)
      gradBias[k] += scale*sums[i];
  }
}

static int cunn_SpatialConvolutionNew_accGradParameters(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  float scale = luaL_optnumber(L, 4, 1);

  luaL_argcheck(L, dW == 1, 1, "dW must be 1 (this will be fixed soon)");
  luaL_argcheck(L, dH == 1, 1, "dH must be 1 (this will be fixed soon)");

  THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *gradBias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");

  float *gradBias_data = THCudaTensor_data(gradBias);
  float *gradOutput_data = THCudaTensor_data(gradOutput);

  if (input->nDimension == 3)
  {
    /* check dims */
    THArgCheck(nOutputPlane == gradOutput->size[0], 1, "Number of output features is not equal to nOutputPlane");

    /* gradient to bias */
    dim3 blocks(nOutputPlane);
    dim3 threads(32);
    compute_gradBias <<<blocks, threads>>> (gradBias_data, gradOutput_data, scale,
                                            gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);

    /* gradient to kernels */
    THCudaTensor_conv2DRevger(gradWeight, 1.0, scale, input, gradOutput, dH, dW);
  }
  else
  {
    /* check dims */
    THArgCheck(nOutputPlane == gradOutput->size[1], 1, "Number of output features is not equal to nOutputPlane");

    /* gradient to bias */
    dim3 blocks(nOutputPlane);
    long sl;
    for (sl=0; sl<gradOutput->size[0]; sl+=16) {
      int cst = 16;
      if ((cst+sl) > gradOutput->size[0]) cst = gradOutput->size[0] - sl;
      dim3 threads(16, cst);
      compute_gradBias <<<blocks, threads>>> (gradBias_data, gradOutput_data + sl*gradOutput->stride[0], scale,
                                              gradOutput->size[1], gradOutput->size[2], gradOutput->size[3]);
    }

    /* gradient to kernels */
    THCudaTensor_conv2DRevgerm(gradWeight, 1.0, scale, input, gradOutput, dH, dW);
  }

  return 0;
}

static const struct luaL_Reg cunn_SpatialConvolutionNew__ [] = {
  {"SpatialConvolutionNew_updateOutput", cunn_SpatialConvolutionNew_updateOutput},
  {"SpatialConvolutionNew_updateGradInput", cunn_SpatialConvolutionNew_updateGradInput},
  {"SpatialConvolutionNew_accGradParameters", cunn_SpatialConvolutionNew_accGradParameters},
  {NULL, NULL}
};

static void cunn_SpatialConvolutionNew_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialConvolutionNew__, "nn");
  lua_pop(L,1);
}
