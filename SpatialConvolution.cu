#include "hip/hip_runtime.h"
#ifndef assert
#define assert(e)  \
    if (!(e)) { \
        printf("failed assertion `%s'\n", #e); \
        THError("aborting..."); \
    };
#endif

/*

This file contains 4 kernels :
- copyPixelsInSlices and its more optimized version copyPixelsInSlicesReg (when there is an upper bound on the number of planes).
- addPixelsInSlices and its optimized version addPixelsInSlicesReg.

The primary kernel is copyPixelsInSlices : it unfolds a 3D matrix into a 2D matrix in a way that the 2D convolution (with many kernels) becomes a matrix multiplication.
We call the resulting matrix "kernelSlices". Each row corresponds to a kW*kH*nInputPlane array.

Steps :
1) choose a pixel (pixi = blockIdx.x, pixj = blockIdx.y)
2) find which slices (coordinates (imin-imax, jmin-jmax)) will contain the pixel information
3) loop : copy the pixel information, jump to next slice (and position) by 
		moving the kernelSlices pointer ptrkslices by stridej = (kH*kW - dW) * nInputPlane

	detailed example : pixel (4,4), kernels of size 5*5, stride dW=1 :
	- 1st slice  : top-left coordinates : (imin,jmin)  . Pixel is in coordinates (4,4, position 25) of the slice.
	- 2nd slice  : top-left coordinates : (imin,jmin+1). Pixel is in coordinates (4,3, position 24) of the slice.
	- 3rd slice  : top-left coordinates : (imin,jmin+2). Pixel is in coordinates (4,2, position 23) of the slice.
	- 4th slice  : top-left coordinates : (imin,jmin+2). Pixel is in coordinates (4,1, position 22) of the slice.
	- 5th slice  : top-left coordinates : (imin,jmin+2). Pixel is in coordinates (4,0, position 21) of the slice.
	- when jmax-jmin slices have been filled, we jump to the next series of slices by 
		moving ptrkslices by stridei = (((size2-jmax+jmin-1)*kH -dH)*kW  + (jmax-jmin+1)*dW)*nInputPlane
	- 1st slice  : top-left coordinates : (imin+1,jmin)  . Pixel is in coordinates (3,4, position 20) of the slice.
	- 2nd slice  : top-left coordinates : (imin+1,jmin+1). Pixel is in coordinates (3,3, position 19) of the slice.
	- 3rd slice  : top-left coordinates : (imin+1,jmin+2). Pixel is in coordinates (3,2, position 18) of the slice.
	- 4th slice  : top-left coordinates : (imin+1,jmin+2). Pixel is in coordinates (3,1, position 17) of the slice.
	- 5th slice  : top-left coordinates : (imin+1,jmin+2). Pixel is in coordinates (3,0, position 16) of the slice.
	- ...

In case the pixel (pixi,pixj) is in the zero-padding, we fill the slice with zeros.

addPixelsInSlices is the same, except we read the contents of the array instead of writing.

the *Reg versions just consist in preloading the pixel information before writing it.


*/

__global__ void copyPixelsInSlices(float *ptrinput, float *ptrkslices,
	int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int nInputPlane2, int valuesperthread, int padleft, int padright, int padup, int paddown)
{
	const int pixi=blockIdx.x;
	const int pixj=blockIdx.y;
	const int blk =blockDim.x;
	const int tidx=threadIdx.x;

        int imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
        int jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
        int imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
        int jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;

	int i;
	int j;
	int k;

	bool zeropad=pixi<padup || pixi>isize1-1+padup || pixj<padleft || pixj>isize2-1+padleft ;
	
	ptrinput   += ((pixi-padup) * isize2 + (pixj-padleft)) * nInputPlane ;
	ptrkslices += ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane2;

	int stridej = (kH*kW - dW) * nInputPlane2;
	int stridei = (((size2-jmax+jmin-1)*kH -dH)*kW  + (jmax-jmin+1)*dW)*nInputPlane2;
	
	if(tidx<nInputPlane2) {
		for(i=imin; i<imax+1; i++) {
			for(j=jmin; j<jmax+1; j++) {
				if(zeropad) 
				{
					for(k=0; k<valuesperthread; k++) {
						ptrkslices[k*blk+tidx]=0;
					}
				}
				else {
					for(k=0; k<valuesperthread; k++) {
						ptrkslices[k*blk+tidx]=ptrinput[k*blk+tidx];
					}
				}
				ptrkslices += stridej;
			}
			ptrkslices += stridei;
		}	
	}
}


__global__ void addPixelsInSlices(float *ptrgradinput, float *ptrkslices,
	int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int valuesperthread, int padleft, int padright, int padup, int paddown)
{
	const int pixi=blockIdx.x;
	const int pixj=blockIdx.y;
	const int blk =blockDim.x;
	const int tidx=threadIdx.x;

        int imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
        int jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
        int imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
        int jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;

	int i;
	int j;
	int k;

	bool zeropad=pixi<padup || pixi>isize1-1+padup || pixj<padleft || pixj>isize2-1+padleft ;
	
	ptrgradinput += ((pixi-padup) * isize2 + (pixj-padleft)) * nInputPlane ;
	ptrkslices   += ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane;

	int stridej = (kH*kW - dW) * nInputPlane;
	int stridei = (((size2-jmax+jmin-1)*kH -dH)*kW  + (jmax-jmin+1)*dW)*nInputPlane;

	for(k=0; k<valuesperthread; k++) {
		ptrgradinput[k*blk+tidx] = 0;
	}
	
	if(tidx<nInputPlane) {
		if(!zeropad) {
			for(i=imin; i<imax+1; i++) {
				for(j=jmin; j<jmax+1; j++) {
						for(k=0; k<valuesperthread; k++) {
							ptrgradinput[k*blk+tidx] += ptrkslices[k*blk+tidx];
						}
					ptrkslices += stridej;
				}
				ptrkslices += stridei;
			}	
		}
	}
}




template <int maxnumplanes> __global__ void addPixelsInSlicesReg(float *ptrgradinput, float *ptrkslices,
	int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int valuesperthread, int padleft, int padright, int padup, int paddown)
{
	const int pixi=blockIdx.x;
	const int pixj=blockIdx.y;
	const int blk =blockDim.x;
	const int tidx=threadIdx.x;

        int imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
        int jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
        int imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
        int jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;

	int i;
	int j;
	int k;

	float gradvalues[maxnumplanes/32];
		for(k=0; k<valuesperthread; k++) {
			gradvalues[k]=0;
		}

	bool zeropad=pixi<padup || pixi>isize1-1+padup || pixj<padleft || pixj>isize2-1+padleft ;
	
	ptrgradinput += ((pixi-padup) * isize2 + (pixj-padleft)) * nInputPlane ;
	ptrkslices   += ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane;

	int stridej = (kH*kW - dW) * nInputPlane;
	int stridei = (((size2-jmax+jmin-1)*kH -dH)*kW  + (jmax-jmin+1)*dW)*nInputPlane;

	if(tidx<nInputPlane) {
		if(!zeropad) {
			for(i=imin; i<imax+1; i++) {
				for(j=jmin; j<jmax+1; j++) {
					for(k=0; k<valuesperthread; k++) {
						gradvalues[k] += ptrkslices[k*blk+tidx];
					}
				ptrkslices += stridej;
				}
				ptrkslices += stridei;
			}	
			for(k=0; k<valuesperthread; k++) {
				ptrgradinput[k*blk+tidx] = gradvalues[k];
			}
		}
	}
}


template <int maxnumplanes> __global__ void copyPixelsInSlicesReg(float *ptrinput, float *ptrkslices,
	int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int valuesperthread, int padleft, int padright, int padup, int paddown)
{
	// each block does one pixel of the input image
	// each kernel slice is represented by its upper-left coordinates

	const int pixi=blockIdx.x;
	const int pixj=blockIdx.y;
	const int blk =blockDim.x;
	const int tidx=threadIdx.x;

	int i,j,k;

	// step 1 : find which kernel slices contain the values of the pixel
        const int imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
        const int jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
        const int imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
        const int jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;

	// step 2 : move the pointers
	// this one goes to where the pixel is at
	ptrinput   += ((pixi-padup) * isize2 + (pixj-padleft)) * nInputPlane ;
	// this one goes to the first pixel of the first kernel slice
	ptrkslices += ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane;

	bool zeropad = pixi<padup || pixi>isize1-1+padup || pixj<padleft || pixj>isize2-1+padleft ;
	// read pixel
	// load the stuff in shared memory first...
	float pixvalues[maxnumplanes/32];
	if(tidx<nInputPlane) {
		if (zeropad) 
		{
			for(k=0; k<valuesperthread; k++) {
				pixvalues[k]=0;
			}
		}
		else
		{
			for(k=0; k<valuesperthread; k++) {
				pixvalues[k]=ptrinput[k*blk+tidx];
			}
		}
	}

	int stridej = (kH*kW - dW) * nInputPlane;
//	int stridei = (((size2-jmax+jmin-1)*kH -dH)*kW  + (jmax-jmin+1)*dW)*nInputPlane;
	int stridei = (size2*kH-dH) * kW *nInputPlane - (jmax-jmin+1) * stridej ;

//	write to memory
	if(tidx<nInputPlane) {
		for(i=imin; i<imax+1; i++) {
			for(j=jmin; j<jmax+1; j++) {
				if(zeropad) 
				{
					for(k=0; k<valuesperthread; k++) {
						ptrkslices[k*blk+tidx]=0;
					}
				}
				else {
					for(k=0; k<valuesperthread; k++) {
						ptrkslices[k*blk+tidx]=pixvalues[k];
					}
				}
				ptrkslices += stridej;
			}
			ptrkslices += stridei;
		}	
	}
}


__global__ void copyPixelsInSlicesRGB(float *ptrinput, float *ptrkslices,
	int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int padleft, int padright, int padup, int paddown)
{
	// each block does one pixel of the input image
	// each kernel slice is represented by its upper-left coordinates

	const int pixi=blockIdx.x;
	const int pixj=blockIdx.y*blockDim.y + threadIdx.y;
	const int tidx=threadIdx.x;

	int i,j;

	if(pixj > isize2 + padleft + padright -1) return;

	// step 1 : find which kernel slices contain the values of the pixel
        const int imin=(pixi - (kH - 1) + (dH -1))/dH > 0 ? (pixi - (kH - 1) + (dH -1))/dH : 0 ;
        const int jmin=(pixj - (kW - 1) + (dW -1))/dW > 0 ? (pixj - (kW - 1) + (dW -1))/dW : 0 ;
        const int imax= pixi / dH < size1 ? pixi / dH : size1 - 1 ;
        const int jmax= pixj / dW < size2 ? pixj / dW : size2 - 1 ;

	// step 2 : move the pointers
	// this one goes to where the pixel is at
	ptrinput   += ((pixi-padup) * isize2 + (pixj-padleft)) * nInputPlane ;
	ptrkslices += ((imin * size2  + jmin) * kH * kW +  (pixi - imin * dH) * kW + (pixj - jmin*dW) ) * nInputPlane;

	bool zeropad = pixi<padup || pixi>isize1-1+padup || pixj<padleft || pixj>isize2-1+padleft ;
	// read pixel
	// load the stuff first...
	float pixvalue;
	if (zeropad) 	{
		pixvalue=0;
	}
	else	{
		pixvalue=ptrinput[tidx];
	}

	int stridej = (kH*kW - dW) * nInputPlane;
	int stridei = (size2*kH-dH) * kW *nInputPlane - (jmax-jmin+1) * stridej ;

//	write to memory
	for(i=imin; i<imax+1; i++) {
		for(j=jmin; j<jmax+1; j++) {
			if(zeropad) 
			{
				ptrkslices[tidx]=0;
			}
			else {
				ptrkslices[tidx]=pixvalue;
			}
			ptrkslices += stridej;
		}
		ptrkslices += stridei;
	}	
}


__global__ void copyBiasToOutputs(float *ptrbias, float *ptroutput, const int size1, const int size2, const int nOutputPlane)
{
	// each thread has a value to manage...
	//const int blk =blockDim.x;
	const int tidx=blockDim.x*blockIdx.x + threadIdx.x;
	const int tidy=blockIdx.y;

	int i;

	float val = ptrbias[tidx];
	ptroutput+= tidy*size1*nOutputPlane;

	for(i=0; i<size1; i++) {
		ptroutput[i*nOutputPlane+tidx]=val;
	}
}



__global__ void computeGradBias32(float *ptrgradbias, float *ptrgradoutput, const int size1, const int size2, const int nOutputPlane, bool add)
{
	const int tid = blockDim.x*blockIdx.x + threadIdx.x;
	const int tidx = threadIdx.x;
	const int tidy = threadIdx.y;
	const int numpix=size1*size2;
	
	__shared__ float values[32][32];

	float value = 0;
	int i;

	for(i=0; i+tidy<numpix; i+=blockDim.y) {
		value += ptrgradoutput[(i+tidy)*nOutputPlane+tid];
	}

	values[tidy][tidx]=value;
	__syncthreads();
	// reduction :

	if (tidy == 0) {
		float gradbiasvalue=0;
		#pragma unroll
		for(i=0; i<32;i++){ gradbiasvalue+=values[i][tidx]; }

		ptrgradbias[tid]=gradbiasvalue;
	}
	
}



static int cunxn_SpatialConvolution_updateOutput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *kernels = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *kernelSlices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "kernelSlices", "torch.CudaTensor");
  long kW = luaT_getfieldcheckint(L, 1, "kW");
  long kH = luaT_getfieldcheckint(L, 1, "kH");
  long dW = luaT_getfieldcheckint(L, 1, "dW");
  long dH = luaT_getfieldcheckint(L, 1, "dH");
  long padup = luaT_getfieldcheckint(L, 1, "padup");
  long paddown = luaT_getfieldcheckint(L, 1, "paddown");
  long padleft = luaT_getfieldcheckint(L, 1, "padleft");
  long padright = luaT_getfieldcheckint(L, 1, "padright");
  long shdmem = luaT_getfieldcheckint(L, 1, "shdmem");
  long nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  long nInputPlane2 = luaT_getfieldcheckint(L, 1, "nInputPlane");

  //luaL_argcheck(L, dimension >= 0 && dimension < input->nDimension, 2, "dimension out of range");

  assert(nInputPlane2%32 == 0 || nInputPlane2<32);
  assert(nOutputPlane%32 == 0);


  // input should be contiguous already but... well.
  // input = THCudaTensor_newContiguous(input);
  long nInputPlane=input->stride[1];
	//printf("%d", nInputPlane);

  // find the size of kernelslices
  long isize1 = input->size[0];
  long isize2 = input->size[1];
  long size1 = (isize1 - kH + padup + paddown) / dH + 1;
  long size2 = (isize2 - kW + padleft + padright) / dW + 1;

//  THCudaTensor* kernelSlices = THCudaTensor_newWithSize1d(size1*size2*kW*kH*nInputPlane);
  THCudaTensor_resize1d(kernelSlices, size1*size2*kW*kH*nInputPlane2);
  THCudaTensor_resize2d(output, size1* size2, nOutputPlane);

  float* ptrkslices = THCudaTensor_data(kernelSlices);
  float* ptroutput  = THCudaTensor_data(output);
  float* ptrinput   = THCudaTensor_data(input);
  float* ptrbias    = THCudaTensor_data(bias);


  // cuda blocks & threads:
  dim3 blocks (isize1 + padup + paddown, isize2 + padleft + padright);
  dim3 threads (32);
  long valuesperthread=nInputPlane2/32;
  if(valuesperthread==0) { valuesperthread=1; } 

	  //with an upper bound on the number of planes, we can be more efficient
	  //kernel unfold inputs
	  if (nInputPlane2 >1024 || shdmem==0) {
	  copyPixelsInSlices<<<blocks, threads>>>(ptrinput, ptrkslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, nInputPlane2, valuesperthread, padleft, padright, padup, paddown);
	  }
	  else if (nInputPlane2 >512) {
		copyPixelsInSlicesReg <1024> <<<blocks, threads>>>(ptrinput, ptrkslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  }
	  else if (nInputPlane2 >384) {
		copyPixelsInSlicesReg <512> <<<blocks, threads>>>(ptrinput, ptrkslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  }
	  else if (nInputPlane2 >256) {
		copyPixelsInSlicesReg <384> <<<blocks, threads>>>(ptrinput, ptrkslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  }
	  else if (nInputPlane2 >128) {
		copyPixelsInSlicesReg <256> <<<blocks, threads>>>(ptrinput, ptrkslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  }
	  else if (nInputPlane2 >32) {
		copyPixelsInSlicesReg <128> <<<blocks, threads>>>(ptrinput, ptrkslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  }
	  else if (nInputPlane2 ==3) {
		  dim3 blocksRGB (isize1 + padup + paddown, (isize2 + padleft + padright+9)/10);
		  dim3 threadsRGB (3,10);
		copyPixelsInSlicesRGB <<<blocksRGB, threadsRGB>>>(ptrinput, ptrkslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, padleft, padright, padup, paddown);
	  }
	  else {
		copyPixelsInSlicesReg <32> <<<blocks, threads>>>(ptrinput, ptrkslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, 1, padleft, padright, padup, paddown);
	  }

  //THCudaTensor_free(input); 



  // fill output with biases
  dim3 blocksbias (nOutputPlane/32, size2);
  dim3 threadsbias (32);
  copyBiasToOutputs<<<blocksbias, threadsbias>>>(ptrbias, ptroutput, size1, size2, nOutputPlane); 



  // unfold conv kernels by resizing
  THCudaTensor_resize2d(kernels, nOutputPlane, kW*kH*nInputPlane2);
  THCudaTensor_transpose(kernels, NULL, 0, 1);
  // put kernelslices in matrix mode
  THCudaTensor_resize2d(kernelSlices, size1*size2,kW*kH*nInputPlane2);

//  printf("sgemm\n");
  // do addmm on output
  THCudaTensor_addmm(output, 1,1, kernelSlices, kernels);
//  printf("sgemm end\n");
//  THCudaTensor_free(kernelSlices); 
  THCudaTensor_transpose(kernels, NULL, 0, 1);


  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in copyPixelsInSlices: %s\n", hipGetErrorString(err));
    THError("aborting");
  }

  THCudaTensor_resize3d(output, size1, size2, nOutputPlane);
 
//  THCudaTensor_resizeAs(kslicestest, kernelSlices);
//  THCudaTensor_copy(kslicestest, kernelSlices);

  // final cut:
  //THCudaTensor_select(output, NULL, dimension, 0);

  return 1;
}





static int cunxn_SpatialConvolution_updateGradInput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  long kW = luaT_getfieldcheckint(L, 1, "kW");
  long kH = luaT_getfieldcheckint(L, 1, "kH");
  long dW = luaT_getfieldcheckint(L, 1, "dW");
  long dH = luaT_getfieldcheckint(L, 1, "dH");
  long padup = luaT_getfieldcheckint(L, 1, "padup");
  long paddown = luaT_getfieldcheckint(L, 1, "paddown");
  long padleft = luaT_getfieldcheckint(L, 1, "padleft");
  long padright = luaT_getfieldcheckint(L, 1, "padright");
  long shdmem = luaT_getfieldcheckint(L, 1, "shdmem");
  long nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  long nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
//  long zeroGradients = luaT_getfieldcheckint(L, 1, "zeroGradients");

//  THCudaTensor *kernelSlices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "kernelSlices", "torch.CudaTensor");
  THCudaTensor *backwardSlices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "backwardSlices", "torch.CudaTensor");

  THCudaTensor *kernels = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
//  THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
//  THCudaTensor *gradBias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");

  

  long isize1 = input->size[0];
  long isize2 = input->size[1];
  long size1 = gradOutput->size[0];
  long size2 = gradOutput->size[1];

  THCudaTensor_resize2d(gradOutput, size1* size2, nOutputPlane);
  THCudaTensor_resize2d(backwardSlices, size1*size2,kW*kH*nInputPlane);

// backprop gradinput into the slices
  THCudaTensor_addmm(backwardSlices, 0, 1, gradOutput, kernels);


// we resize gradOutput back to what it was...
  THCudaTensor_resize3d(gradOutput, size1, size2, nOutputPlane);




  THCudaTensor_resizeAs(gradInput, input);

  float* ptrbackslices = THCudaTensor_data(backwardSlices);
  float* ptrgradinput  = THCudaTensor_data(gradInput);

  dim3 blocks (isize1 + padup + paddown, isize2 + padleft + padright);
  dim3 threads (32);
  long valuesperthread=nInputPlane/32;

  if(valuesperthread==0) { valuesperthread=1; } 
  // this is for the specific case of the inputs with less than 32 channels
  // for some reason i thought it would be cool to be able to backprop through it

	  if (nInputPlane >1024 || shdmem==0) {
	  addPixelsInSlices<<<blocks, threads>>>(ptrgradinput, ptrbackslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  }
	  else if (nInputPlane >512)  {
	  addPixelsInSlicesReg <1024> <<<blocks, threads>>>(ptrgradinput, ptrbackslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  } 
	  else if (nInputPlane >384)  {
	  addPixelsInSlicesReg <512> <<<blocks, threads>>>(ptrgradinput, ptrbackslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  } 
	  else if (nInputPlane >256)  {
	  addPixelsInSlicesReg <384> <<<blocks, threads>>>(ptrgradinput, ptrbackslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  } 
	  else if (nInputPlane >128)  {
	  addPixelsInSlicesReg <256> <<<blocks, threads>>>(ptrgradinput, ptrbackslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  } 
	  else if (nInputPlane >32)  {
	  addPixelsInSlicesReg <128> <<<blocks, threads>>>(ptrgradinput, ptrbackslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  } 
	  else {
	  addPixelsInSlicesReg <32> <<<blocks, threads>>>(ptrgradinput, ptrbackslices,
		dH, dW, kH, kW, size1, size2, isize1, isize2, nInputPlane, valuesperthread, padleft, padright, padup, paddown);
	  } 

//  THCudaTensor_copy(kslicestest, kernelSlices);




  return 1;
}



static int cunxn_SpatialConvolution_accGradParameters(lua_State *L)
{
//  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  long kW = luaT_getfieldcheckint(L, 1, "kW");
  long kH = luaT_getfieldcheckint(L, 1, "kH");
//  long dW = luaT_getfieldcheckint(L, 1, "dW");
//  long dH = luaT_getfieldcheckint(L, 1, "dH");
//  long padup = luaT_getfieldcheckint(L, 1, "padup");
//  long paddown = luaT_getfieldcheckint(L, 1, "paddown");
//  long padleft = luaT_getfieldcheckint(L, 1, "padleft");
//  long padright = luaT_getfieldcheckint(L, 1, "padright");
//  long shdmem = luaT_getfieldcheckint(L, 1, "shdmem");
  long nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  long nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
  long zeroGradients = luaT_getfieldcheckint(L, 1, "zeroGradients");

  THCudaTensor *kernelSlices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "kernelSlices", "torch.CudaTensor");

//  THCudaTensor *kernels = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *gradBias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");

//  printf("accgradparameters");

//  long isize1 = input->size[0];
//  long isize2 = input->size[1];
  long size1 = gradOutput->size[0];
  long size2 = gradOutput->size[1];

  THCudaTensor_resize2d(gradOutput, size1* size2, nOutputPlane);

  float* ptrgradbias = THCudaTensor_data(gradBias);
  float* ptrgradoutput  = THCudaTensor_data(gradOutput);
  dim3 blocksgradbias (nOutputPlane/32);
  dim3 threadsgradbias (32,32);

  THCudaTensor_resize2d(gradWeight, nOutputPlane, kW*kH*nInputPlane);
//  THCudaTensor_transpose(gradWeight, NULL, 0, 1);
  THCudaTensor_transpose(gradOutput, NULL, 0, 1);

/*long gwsize1 = gradWeight->size[0];
long gwsize2 = gradWeight->size[1];

long kslsize1 = kernelSlices->size[0];
long kslsize2 = kernelSlices->size[1];

printf("gwsize : %d, %d \n", gwsize1, gwsize2);
printf("kslsize : %d, %d \n", kslsize1, kslsize2);
printf("goutsize : %d, %d \n", size1, size2);*/

  if (zeroGradients == 1) { 
	THCudaTensor_addmm(gradWeight, 0, 1, gradOutput, kernelSlices); 
	computeGradBias32 <<<blocksgradbias, threadsgradbias>>>  (ptrgradbias, ptrgradoutput, size1, size2, nOutputPlane, 0);
  } else {
	THCudaTensor_addmm(gradWeight, 1, 1, gradOutput, kernelSlices); 
	computeGradBias32 <<<blocksgradbias, threadsgradbias>>>  (ptrgradbias, ptrgradoutput, size1, size2, nOutputPlane, 1);
  }  
  THCudaTensor_transpose(gradOutput, NULL, 0, 1);
//  THCudaTensor_transpose(gradWeight, NULL, 0, 1);

// we resize gradOutput back to what it was...
  THCudaTensor_resize3d(gradOutput, size1, size2, nOutputPlane);

return 1;

}

static const struct luaL_Reg cunxn_SpatialConvolution__ [] = {
  {"SpatialConvolution_updateOutput", cunxn_SpatialConvolution_updateOutput},
  {"SpatialConvolution_updateGradInput", cunxn_SpatialConvolution_updateGradInput},
  {"SpatialConvolution_accGradParameters", cunxn_SpatialConvolution_accGradParameters},
  {NULL, NULL}
};

static void cunxn_SpatialConvolution_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunxn_SpatialConvolution__, "nxn");
  lua_pop(L,1);
}
